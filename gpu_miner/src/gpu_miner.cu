#include "hip/hip_runtime.h"
// Copyright Theodor-Ioan Rolea, 333CA, 2024
#include <stdio.h>
#include <stdint.h>
#include "../include/utils.cuh"
#include <string.h>
#include <stdlib.h>
#include <inttypes.h>

// CUDA implementation for strcat
__device__ void d_strcat(char* dest, const char* src) {
    int dest_len = 0;
    while (dest[dest_len] != '\0') {
        dest_len++;
    }
    int src_len = 0;
    while ((dest[dest_len + src_len] = src[src_len]) != '\0') {
        src_len++;
    }
}

__global__ void findNonce(BYTE* prev_block_hash, BYTE* top_hash, BYTE* block_hash, BYTE* difficulty, uint64_t* found_nonce, uint64_t *ok) {
	// Check if nonce has been found
	if (ok[0] == 1) return;

	// Calculate nonce
	uint64_t nonce = blockIdx.x * blockDim.x + threadIdx.x + 1;

	// Check if nonce is valid
    if (nonce > MAX_NONCE) return;

	// Initialize block content
    BYTE block_content[BLOCK_SIZE];
    d_strcpy((char*)block_content, (const char*)prev_block_hash);
    d_strcat((char*)block_content, (const char*)top_hash);

    char nonce_string[NONCE_SIZE];
    intToString(nonce, nonce_string);
    d_strcat((char*)block_content, nonce_string);

	// Calculate hash
    BYTE temp_hash[SHA256_HASH_SIZE];
    apply_sha256(block_content, d_strlen((const char*)block_content), temp_hash, 1);


	// Check if hash is valid
    if (compare_hashes(temp_hash, difficulty) <= 0) {
		// Using lock to prevent
		// multiple threads from writing to the same memory location
        atomicExch((unsigned long long*)found_nonce, (unsigned long long)nonce);

		// Set ok to 1 to stop other threads
		// once nonce has been found
		ok[0] = 1;

		// Copy hash to block_hash
        d_strcpy((char*)block_hash, (const char*)temp_hash);
    }
}

int main(int argc, char **argv) {
	BYTE hashed_tx1[SHA256_HASH_SIZE], hashed_tx2[SHA256_HASH_SIZE], hashed_tx3[SHA256_HASH_SIZE], hashed_tx4[SHA256_HASH_SIZE],
			tx12[SHA256_HASH_SIZE * 2], tx34[SHA256_HASH_SIZE * 2], hashed_tx12[SHA256_HASH_SIZE], hashed_tx34[SHA256_HASH_SIZE],
			tx1234[SHA256_HASH_SIZE * 2], top_hash[SHA256_HASH_SIZE], block_content[BLOCK_SIZE];
	BYTE block_hash[SHA256_HASH_SIZE] = "0000000000000000000000000000000000000000000000000000000000000000";
	BYTE *d_prev_block_hash, *d_top_hash, *d_block_hash, *d_difficulty;
    uint64_t *d_found_nonce;

	// Top hash
	apply_sha256(tx1, strlen((const char*)tx1), hashed_tx1, 1);
	apply_sha256(tx2, strlen((const char*)tx2), hashed_tx2, 1);
	apply_sha256(tx3, strlen((const char*)tx3), hashed_tx3, 1);
	apply_sha256(tx4, strlen((const char*)tx4), hashed_tx4, 1);
	strcpy((char *)tx12, (const char *)hashed_tx1);
	strcat((char *)tx12, (const char *)hashed_tx2);
	apply_sha256(tx12, strlen((const char*)tx12), hashed_tx12, 1);
	strcpy((char *)tx34, (const char *)hashed_tx3);
	strcat((char *)tx34, (const char *)hashed_tx4);
	apply_sha256(tx34, strlen((const char*)tx34), hashed_tx34, 1);
	strcpy((char *)tx1234, (const char *)hashed_tx12);
	strcat((char *)tx1234, (const char *)hashed_tx34);
	apply_sha256(tx1234, strlen((const char*)tx34), top_hash, 1);

	strcpy((char*)block_content, (const char*)prev_block_hash);
	strcat((char*)block_content, (const char*)top_hash);
    
	// Initialize gpu fields
    hipMalloc(&d_prev_block_hash, SHA256_HASH_SIZE);
    hipMalloc(&d_top_hash, SHA256_HASH_SIZE);
    hipMalloc(&d_block_hash, SHA256_HASH_SIZE);
    hipMalloc(&d_difficulty, SHA256_HASH_SIZE);
    hipMalloc(&d_found_nonce, sizeof(uint64_t));

    uint64_t found_nonce = 0;
    
	// Copy data to gpu memory
    hipMemcpy(d_prev_block_hash, prev_block_hash, SHA256_HASH_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_top_hash, top_hash, SHA256_HASH_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_block_hash, block_hash, SHA256_HASH_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_difficulty, difficulty_5_zeros, SHA256_HASH_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_found_nonce, &found_nonce, sizeof(uint64_t), hipMemcpyHostToDevice);

	// Initialize block and threads
    int blockSize = 256;
    int numBlocks = (MAX_NONCE / blockSize) + 1;

	// Initialize the ok flag
	int ok[1] = {0};
	uint64_t *d_ok;
	hipMalloc(&d_ok, sizeof(uint64_t));
	hipMemcpy(d_ok, ok, sizeof(uint64_t), hipMemcpyHostToDevice);

	hipEvent_t start, stop;
    startTiming(&start, &stop);
    
    findNonce<<<numBlocks, blockSize>>>(d_prev_block_hash, d_top_hash, d_block_hash, d_difficulty, d_found_nonce, d_ok);

    hipDeviceSynchronize();

	float seconds = stopTiming(&start, &stop);

	// Copy data from gpu memory
    hipMemcpy(block_hash, d_block_hash, SHA256_HASH_SIZE, hipMemcpyDeviceToHost);
    hipMemcpy(&found_nonce, d_found_nonce, sizeof(uint64_t), hipMemcpyDeviceToHost);
    
	printResult(block_hash, found_nonce, seconds);

	// Free everything
    hipFree(d_prev_block_hash);
    hipFree(d_top_hash);
    hipFree(d_block_hash);
    hipFree(d_difficulty);
    hipFree(d_found_nonce);

	return 0;
}
